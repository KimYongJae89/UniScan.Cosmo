#include "hip/hip_runtime.h"
#include "DeviceRansacFunction.cuh"
#include <hiprand/hiprand_kernel.h>

__device__ int lock;

__device__ int kernel_GetCost(double* xArray, double* yArray, unsigned int size, double gradient, double centerX, double centerY, double distThreshold)
{
	double cost = 0;

	if (gradient != 0)
	{
		double lowerSide = sqrt(pow(gradient, 2) + 1);

		for (int i = 0; i < size; i++)
		{
			if (abs(((xArray[i] - centerX) * gradient) + centerY - yArray[i]) / lowerSide <= distThreshold)
				cost++;
		}
	}

	return cost;
}

__global__ void kernel_Compute(double* xArray, double* yArray, double* gradientArray, double* centerXArray, double* centerYArray, unsigned int* costArray, unsigned int size, int iter, double distThreshold, unsigned long seed)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= iter)
		return;

	hiprandState state;
	hiprand_init(seed, id, 0, &state);

	int randNum1 = (int)(hiprand_uniform(&state) * (float)(size - 1));
	int randNum2 = (int)(hiprand_uniform(&state) * (float)(size - 1));

	double x1 = xArray[randNum1];
	double y1 = yArray[randNum1];

	double x2 = xArray[randNum2];
	double y2 = yArray[randNum2];

	centerXArray[id] = (x1 + x2) / 2.0;
	centerYArray[id] = (y1 + y2) / 2.0;

	if (x1 == x2)
		gradientArray[id] = DBL_MAX;
	else
		gradientArray[id] = (y2 - y1) / (x2 - x1);

	int cost = kernel_GetCost(xArray, yArray, size, gradientArray[id], centerXArray[id], centerYArray[id], distThreshold);
	
	costArray[id] = cost;
}